
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/
int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < dim && row < dim) {
        int index = row * dim + col;
        C[index] = A[index] + B[index];
    } }  


 //int c = threadIdx.x + blockDim.x * blockIdx.x;
    //if (c<dim*dim) C[c] = A[c] + B[c];}
void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------
// Launch the kernel
   
const unsigned int BLOCK_SIZE = TILE_SIZE;
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((dim + dimBlock.x - 1) / dimBlock.x, (dim + dimBlock.y - 1) / dimBlock.y);

    matAdd<<<dimGrid, dimBlock>>>(dim, A, B, C);
// Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	
    /*************************************************************************/
    hipDeviceSynchronize();
}
